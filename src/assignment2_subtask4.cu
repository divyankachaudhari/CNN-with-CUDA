#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <cfloat>
#include <cmath>
#include <fstream>
#include <string>
#include <hipblas.h>
#include <sstream>
#include <thrust/device_vector.h>
#include <thrust/sort.h>
#include <thrust/execution_policy.h>
#include <thrust/pair.h>
#include <algorithm> // For std::sort
#include <iostream>
#include <chrono>
#include <boost/filesystem.hpp>

namespace fs = boost::filesystem;

void printFlatMatrix(const std::vector<float>& matrix, int rows, int cols) {
    for (int i = 0; i < rows; ++i) {
        for (int j = 0; j < cols; ++j) {
            std::cout << matrix[i * cols + j] << " ";
        }
        std::cout << std::endl;
    }
}

void loadMNISTImageMatrix(const std::string& filename, std::vector<float>& image) {
    std::ifstream file(filename);
    std::string line;
    int row = 0;
    while (std::getline(file, line)) {
        std::stringstream ss(line);
        float pixel;
        int col = 0;
        while (ss >> pixel) {
            image[row * 28 + col] = pixel;
            col++;
        }
        row++;
    }
}


void loadWeightsAndBiases(const std::string& filename, std::vector<float>& weights, std::vector<float>& biases, int biasCount) {
    std::ifstream file(filename);
    if (!file) {
        std::cerr << "Failed to open file: " << filename << std::endl;
        return; // Early return or handle the error as needed
    }
    
    float value;
    while (file >> value) {
        weights.push_back(value);
    }

    // Check if we have enough data for the expected biasCount
    if (weights.size() < static_cast<size_t>(biasCount)) {
        std::cerr << "Not enough data in file for the expected number of biases." << std::endl;
        return; // Handle error appropriately
    }

    biases.insert(biases.end(), weights.end() - biasCount, weights.end());
    weights.erase(weights.end() - biasCount, weights.end());
}


__global__ void addBiasesKernel(float* output, const float* biases, int width, int height, int channels) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;

    if (x < width && y < height && z < channels) {
        int index = z * width * height + y * width + x;
        output[index] += biases[z];
    }
}

__global__ void convMultiChannelKernel(
    const float* input, 
    const float* kernels, 
    float* output, 
    int inputHeight, 
    int inputWidth, 
    int kernelSize, 
    int numInputChannels, 
    int numOutputChannels) {
    
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z; // Output channel

    if (x < inputWidth - kernelSize + 1 && y < inputHeight - kernelSize + 1 && z < numOutputChannels) {
        float sum = 0.0f;

        // Iterate over each input channel
        for (int inCh = 0; inCh < numInputChannels; ++inCh) {
            float channelSum = 0.0f; // Sum for the current input channel
            // Apply the kernel for the current input channel and output channel
            for (int ki = 0; ki < kernelSize; ++ki) {
                for (int kj = 0; kj < kernelSize; ++kj) {
                    int inputIdx = (inCh * inputHeight + y + ki) * inputWidth + x + kj;
                    int kernelIdx = ((z * numInputChannels + inCh) * kernelSize + ki) * kernelSize + kj;
                    // int kernelIdx = ki * kernelSize + kj; // If the same kernel is applied to all input channels

                    channelSum += input[inputIdx] * kernels[kernelIdx];

                    if (x == 0 && y == 0 && z == 0) { // Extended debugging for the first output element
                        // printf("InCh=%d, Ki=%d, Kj=%d: inputIdx=%d, kernelIdx=%d, input=%f, kernel=%f, channelSum=%f\n",
                        //        inCh, ki, kj, inputIdx, kernelIdx, input[inputIdx], kernels[kernelIdx], channelSum);
                    }
                }
            }
            sum += channelSum; // Add the sum from the current channel to the total sum

            if (x == 0 && y == 0 && z == 0) { // Debug each channel's contribution
                // printf("Channel %d sum: %f\n", inCh, channelSum);
            }
        }

        int outputIdx = (z * (inputHeight - kernelSize + 1) + y) * (inputWidth - kernelSize + 1) + x;
        output[outputIdx] = sum;

        // if (x == 0 && y == 0 && z == 0) { // Final output for the first element
        //     printf("Final output: outputIdx=%d, output=%f\n", outputIdx, output[outputIdx]);
        // }
    }
}

void runConvolutionAndAddBiases(const float* input, const float* kernels, float* output, 
                    int inputHeight, int inputWidth, 
                    int kernelSize, int numInputChannels, int numOutputChannels, const float *biases) {
    // Device memory pointers
    float *d_input, *d_kernels, *d_output, *d_biases;

    // Calculate sizes
    size_t inputSize = inputHeight * inputWidth * numInputChannels * sizeof(float);
    size_t kernelSizeTotal = kernelSize * kernelSize * numInputChannels * numOutputChannels * sizeof(float);
    size_t outputWidth = inputWidth - kernelSize + 1;
    size_t outputSize = (inputHeight - kernelSize + 1) * (inputWidth - kernelSize + 1) * numOutputChannels * sizeof(float);

    // Allocate memory on the device
    hipMalloc(&d_input, inputSize);
    hipMalloc(&d_kernels, kernelSizeTotal);
    hipMalloc(&d_output, outputSize);
    hipMalloc(&d_biases, numOutputChannels * sizeof(float));


    // Copy data to the device
    hipMemcpy(d_input, input, inputSize, hipMemcpyHostToDevice);
    hipMemcpy(d_kernels, kernels, kernelSizeTotal, hipMemcpyHostToDevice);
    hipMemcpy(d_biases, biases, numOutputChannels * sizeof(float), hipMemcpyHostToDevice);


    // Define grid and block dimensions
    dim3 threadsPerBlock(16, 16, 1);
    dim3 numBlocks((inputWidth - kernelSize + 1 + 15) / 16, (inputHeight - kernelSize + 1 + 15) / 16, numOutputChannels);

    // Launch the kernel
    hipMemset(d_output, 0, outputSize);
    convMultiChannelKernel<<<numBlocks, threadsPerBlock>>>(d_input, d_kernels, d_output, inputHeight, inputWidth, kernelSize, numInputChannels, numOutputChannels);
    hipDeviceSynchronize();


    addBiasesKernel<<<numBlocks, threadsPerBlock>>>(d_output, d_biases, outputWidth, outputWidth, numOutputChannels);
    // hipDeviceSynchronize();

    // Copy the result back to host
    hipMemcpy(output, d_output, outputSize, hipMemcpyDeviceToHost);


    // Free device memory
    hipFree(d_input);
    hipFree(d_kernels);
    hipFree(d_output);
    hipFree(d_biases);
}


void runConvolution(const float* input, const float* kernels, float* output, 
                    int inputHeight, int inputWidth, 
                    int kernelSize, int numInputChannels, int numOutputChannels) {
    // Device memory pointers
    float *d_input, *d_kernels, *d_output;

    // Calculate sizes
    size_t inputSize = inputHeight * inputWidth * numInputChannels * sizeof(float);
    size_t kernelSizeTotal = kernelSize * kernelSize * numInputChannels * numOutputChannels * sizeof(float);
    size_t outputSize = (inputHeight - kernelSize + 1) * (inputWidth - kernelSize + 1) * numOutputChannels * sizeof(float);

    // Allocate memory on the device
    hipMalloc(&d_input, inputSize);
    hipMalloc(&d_kernels, kernelSizeTotal);
    hipMalloc(&d_output, outputSize);

    // Copy data to the device
    hipMemcpy(d_input, input, inputSize, hipMemcpyHostToDevice);
    hipMemcpy(d_kernels, kernels, kernelSizeTotal, hipMemcpyHostToDevice);

    // Define grid and block dimensions
    dim3 threadsPerBlock(16, 16, 1);
    dim3 numBlocks((inputWidth - kernelSize + 1 + 15) / 16, (inputHeight - kernelSize + 1 + 15) / 16, numOutputChannels);

    // Launch the kernel
    hipMemset(d_output, 0, outputSize);
    convMultiChannelKernel<<<numBlocks, threadsPerBlock>>>(d_input, d_kernels, d_output, inputHeight, inputWidth, kernelSize, numInputChannels, numOutputChannels);

    // Copy the result back to host
    hipMemcpy(output, d_output, outputSize, hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_input);
    hipFree(d_kernels);
    hipFree(d_output);
}

void addBiasesToConvolutionOutput(float* output, const float* biases, int outputHeight, int outputWidth, int numOutputChannels) {
    // Allocate device memory for the biases
    float *d_biases;
    hipMalloc(&d_biases, numOutputChannels * sizeof(float));
    hipMemcpy(d_biases, biases, numOutputChannels * sizeof(float), hipMemcpyHostToDevice);

    // Set up the grid and block dimensions
    dim3 threadsPerBlock(16, 16, 1);
    dim3 numBlocks((outputWidth + 15) / 16, (outputHeight + 15) / 16, numOutputChannels);

    // Launch the kernel
    addBiasesKernel<<<numBlocks, threadsPerBlock>>>(output, d_biases, outputWidth, outputHeight, numOutputChannels);

    // Free device memory for biases
    hipFree(d_biases);
}

__global__ void maxPoolingKernel(const float *input, float *output, int inputHeight, int inputWidth, int outputHeight, int outputWidth, int numChannels, int stride, int poolSize) {
    int outX = blockIdx.x * blockDim.x + threadIdx.x;
    int outY = blockIdx.y * blockDim.y + threadIdx.y;
    int channel = blockIdx.z * blockDim.z + threadIdx.z;

    if (outX < outputWidth && outY < outputHeight && channel < numChannels) {
        float maxVal = -FLT_MAX;
        for (int poolY = 0; poolY < poolSize; ++poolY) {
            for (int poolX = 0; poolX < poolSize; ++poolX) {
                int inX = outX * stride + poolX;
                int inY = outY * stride + poolY;
                if (inX < inputWidth && inY < inputHeight) {
                    int idx = channel * (inputHeight * inputWidth) + inY * inputWidth + inX;
                    maxVal = fmaxf(maxVal, input[idx]);
                }
            }
        }
        int outIdx = channel * (outputHeight * outputWidth) + outY * outputWidth + outX;
        output[outIdx] = maxVal;
    }
}

__global__ void reluKernel(float* data, int count) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < count) {
        data[index] = max(0.0f, data[index]);
    }
}

__global__ void softmaxKernel(float* input, float* output, int count) {
    extern __shared__ float sharedData[];
    int tid = threadIdx.x;
    float maxVal = -FLT_MAX;

    // Find max value for numerical stability
    for (int i = 0; i < count; ++i) {
        if (input[i] > maxVal) {
            maxVal = input[i];
        }
    }
    
    // Load input into shared memory, taking exponential
    if (tid < count) {
        sharedData[tid] = exp(input[tid] - maxVal); // Improve numerical stability
    }
    __syncthreads();

    // Compute the sum of all exponentials
    float sum = 0.0f;
    for (int i = 0; i < count; ++i) {
        sum += sharedData[i];
    }

    // Normalize
    if (tid < count) {
        output[tid] = sharedData[tid] / sum;
    }
}

int main(int argc, char* argv[]) {
    const fs::path directoryPath("pre-proc-img");
    // STart clock


    // Assuming 28x28 input image size (MNIST)
    std::vector<float> image(28 * 28); // Assuming MNIST images are 28x28 pixels
    const int imageSize = 28;
    // Load weights and biases for the first convolutional layer
    std::vector<float> weightsConv1, biasesConv1;
    loadWeightsAndBiases("weights/conv1.txt", weightsConv1, biasesConv1, 20); // Assuming 
        // Load weights and biases for the second fully connected layer
    // Load weights and biases for the second convolutional layer
    std::vector<float> weightsConv2, biasesConv2;
    loadWeightsAndBiases("weights/conv2.txt", weightsConv2, biasesConv2, 50); // Assuming 50 biases for Conv2 layer

        // Load weights and biases for the first fully connected layer
    std::vector<float> weightsFC1, biasesFC1;
    loadWeightsAndBiases("weights/fc1.txt", weightsFC1, biasesFC1, 500); // Assuming 500 biases for FC1 layer

    std::vector<float> weightsFC2, biasesFC2;
    loadWeightsAndBiases("weights/fc2.txt", weightsFC2, biasesFC2, 10); // Assuming 10 biases for FC2 layer

    std::vector<float> outputConv1(24 * 24 * 20);

    // Allocate memory for the output of the max pooling operation
    // Input: d_outputConv1, Output: d_pooledOutputConv1
    float *d_pooledInputConv1, *d_pooledOutputConv1; 
    hipMalloc(&d_pooledOutputConv1, 12 * 12 * 20 * sizeof(float));
    hipMalloc(&d_pooledInputConv1, 24 * 24 * 20 * sizeof(float));


    // Define kernel execution configuration for maxPoolingKernel
    dim3 threadsPerBlock(16, 16, 1); // Using 1 for z-dimension since pooling is applied per channel
    dim3 numBlocksPoolingConv1(
    (12 + threadsPerBlock.x - 1) / threadsPerBlock.x,
    (12 + threadsPerBlock.y - 1) / threadsPerBlock.y,
    20); // One block per channel

    // Prepare output array for the convolution result
    std::vector<float> outputConv2(8 * 8 * 50);
    // Allocate memory for the output of the max pooling operation
    // Input: d_outputConv2, Output: d_pooledOutputConv2
    float *d_pooledInputConv2, *d_pooledOutputConv2;
    hipMalloc(&d_pooledOutputConv2, 4 * 4 * 50 * sizeof(float));
    hipMalloc(&d_pooledInputConv2, 8 * 8 * 50 * sizeof(float));

    // Define kernel execution configuration for maxPoolingKernel
    // dim3 threadsPerBlock(16, 16, 1); // Using 1 for z-dimension since pooling is applied per channel
    dim3 numBlocksPoolingConv2(
        (4 + threadsPerBlock.x - 1) / threadsPerBlock.x,
        (4 + threadsPerBlock.y - 1) / threadsPerBlock.y,
        50); // One block per channel


    const int inputHeightFC1 = 4; // 4
    const int inputWidthFC1 = 4; // 4
    const int numChannelsFC1 = 50; // 50
    const int kernelSizeFC1 = 4; // 4

    const int outputHeightFC1 = 1; // 1
    const int outputWidthFC1 = 1; // 1
    const int numOutputChannelsFC1 = 500; // 500


    // Prepare output array for the convolution result
    std::vector<float> outputFC1(outputHeightFC1 * outputWidthFC1 * numOutputChannelsFC1);


    // After the convolution, add biases
    float* d_outputFC1;
    float* d_biasesFC1;
    hipMalloc(&d_outputFC1, outputFC1.size() * sizeof(float));
    hipMalloc(&d_biasesFC1, biasesFC1.size() * sizeof(float));


    // dim3 threadsPerBlock(16, 16, 1);
    dim3 numBlocksFC1((outputWidthFC1 + 15) / 16, (outputHeightFC1 + 15) / 16, numOutputChannelsFC1);
    // Assume outputFC1 holds the FC1 output data on the device
    int totalOutputCountFC1 = outputHeightFC1 * outputWidthFC1 * numOutputChannelsFC1; // For FC1, this is essentially 500
    dim3 threadsPerBlockReLU(256);
    dim3 blocksPerGridReLU((totalOutputCountFC1 + threadsPerBlockReLU.x - 1) / threadsPerBlockReLU.x);
    const int inputHeightFC2 = outputHeightFC1; // 1 
    const int inputWidthFC2 = outputWidthFC1; // 1
    const int numChannelsFC2 = numOutputChannelsFC1; // 500
    const int kernelSizeFC2 = 1; // 1

    const int outputHeightFC2 = 1; // 1
    const int outputWidthFC2 = 1; // 1
    const int numOutputChannelsFC2 = 10; // 10


    // Prepare output array for the convolution result
    std::vector<float> outputFC2(outputHeightFC2 * outputWidthFC2 * numOutputChannelsFC2);

    // After the convolution, add biases
    float* d_outputFC2;
    float* d_biasesFC2;
    hipMalloc(&d_outputFC2, outputFC2.size() * sizeof(float));
    hipMalloc(&d_biasesFC2, biasesFC2.size() * sizeof(float));
    // dim3 threadsPerBlock(16, 16, 1);
    dim3 numBlocksFC2((outputWidthFC2 + 15) / 16, (outputHeightFC2 + 15) / 16, numOutputChannelsFC2);


        // Assume outputFC2 has been computed and contains raw logits from FC2
    int numElementsFC2 = numOutputChannelsFC2; // For softmax, this is 10
    std::vector<float> softmaxOutput(numElementsFC2);

    // Allocate memory for softmax output on device
    float *d_softmaxOutput;
    hipMalloc(&d_softmaxOutput, numElementsFC2 * sizeof(float));

    int threadsPerBlockSoftMax = 256; // Can be tuned
    int sharedDataSize = numElementsFC2 * sizeof(float); // Required shared memory



    auto start = std::chrono::high_resolution_clock::now();
    if (fs::exists(directoryPath) && fs::is_directory(directoryPath)) {
        fs::directory_iterator end_itr; // Default construction yields past-the-end
        for (fs::directory_iterator itr(directoryPath); itr != end_itr; ++itr) {
            if (fs::is_regular_file(itr->status())) {
                const auto& filePath = itr->path();
                if (filePath.extension() == ".txt") {// Check if the file is a text file
            // std::cout << "Processing " << filePath << std::endl;
            std::string filename = filePath.filename().string();

  
            loadMNISTImageMatrix(filePath.string(), image);
            

// ---------------- First Convolution Layer ----------------------//

    runConvolutionAndAddBiases(image.data(), weightsConv1.data(), outputConv1.data(), 
                   imageSize, imageSize, 5, 1, 20, biasesConv1.data());

// ---------------- Max Pooling for Conv1 ----------------------// 


    hipMemcpy(d_pooledInputConv1, outputConv1.data(), outputConv1.size() * sizeof(float), hipMemcpyHostToDevice);


    // __global__ void maxPoolingKernel(const float *input, float *output, int inputHeight, int inputWidth, int outputHeight, int outputWidth, int numChannels, int stride, int poolSize)
    maxPoolingKernel<<<numBlocksPoolingConv1, threadsPerBlock>>>(d_pooledInputConv1, d_pooledOutputConv1, 24, 24, 12, 12, 20, 2, 2);
    hipDeviceSynchronize();

    // Check for any errors launching the kernel
    hipError_t poolingError = hipGetLastError();
    if (poolingError != hipSuccess) {
        std::cerr << "CUDA error in maxPoolingKernel: " << hipGetErrorString(poolingError) << std::endl;
    }

    // Example: Copy the pooled output back to the host for inspection
    std::vector<float> pooledOutputConv1(12 * 12 * 20);
    hipMemcpy(pooledOutputConv1.data(), d_pooledOutputConv1, pooledOutputConv1.size() * sizeof(float), hipMemcpyDeviceToHost);



 // ----------------- Second Convolution Layer ----------------- //

    runConvolutionAndAddBiases(pooledOutputConv1.data(), weightsConv2.data(), outputConv2.data(), 
                   12, 12, 5, 20, 50, biasesConv2.data());



    // ---------------- Max Pooling for Conv2 ----------------------//

    hipMemcpy(d_pooledInputConv2, outputConv2.data(), outputConv2.size() * sizeof(float), hipMemcpyHostToDevice);


    // __global__ void maxPoolingKernel(const float *input, float *output, int inputHeight, int inputWidth, int outputHeight, int outputWidth, int numChannels, int stride, int poolSize)
    maxPoolingKernel<<<numBlocksPoolingConv2, threadsPerBlock>>>(d_pooledInputConv2, d_pooledOutputConv2, 8, 8, 4, 4, 50, 2, 2);
    hipDeviceSynchronize();

    // Check for any errors launching the kernel
    hipError_t poolingErrorConv2 = hipGetLastError();
    if (poolingErrorConv2 != hipSuccess) {
        std::cerr << "CUDA error in maxPoolingKernel: " << hipGetErrorString(poolingErrorConv2) << std::endl;
    }

    // Example: Copy the pooled output back to the host for inspection
    std::vector<float> pooledOutputConv2(4 * 4 * 50);
    hipMemcpy(pooledOutputConv2.data(), d_pooledOutputConv2, pooledOutputConv2.size() * sizeof(float), hipMemcpyDeviceToHost);

// ----------------- Fully Connected Layer 1 ----------------- //

    // Run convolution on GPU
    runConvolution(pooledOutputConv2.data(), weightsFC1.data(), outputFC1.data(), 
                   inputHeightFC1, inputWidthFC1, kernelSizeFC1, numChannelsFC1, numOutputChannelsFC1);
    // runConvolutionAndAddBiases(pooledOutputConv2.data(), weightsFC1.data(), outputFC1.data(), 
    //                inputHeightFC1, inputWidthFC1, kernelSizeFC1, numChannelsFC1, numOutputChannelsFC1, biasesFC1.data());
    hipDeviceSynchronize();

    hipMemcpy(d_outputFC1, outputFC1.data(), outputFC1.size() * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_biasesFC1, biasesFC1.data(), biasesFC1.size() * sizeof(float), hipMemcpyHostToDevice);

    addBiasesKernel<<<numBlocksFC1, threadsPerBlock>>>(d_outputFC1, d_biasesFC1, outputWidthFC1, outputHeightFC1, numOutputChannelsFC1); 
    hipDeviceSynchronize();

    // hipMemcpy(d_outputFC1, outputFC1.data(), outputFC1.size() * sizeof(float), hipMemcpyDeviceToHost);

    reluKernel<<<blocksPerGridReLU, threadsPerBlockReLU>>>(d_outputFC1, totalOutputCountFC1);
    hipDeviceSynchronize();


    // Copy back the result after adding biases
    hipMemcpy(outputFC1.data(), d_outputFC1, outputFC1.size() * sizeof(float), hipMemcpyDeviceToHost);



// ----------------- Fully Connected Layer 2 ----------------- //

    // Run convolution on GPU
    runConvolution(outputFC1.data(), weightsFC2.data(), outputFC2.data(), 
                   inputHeightFC2, inputWidthFC2, kernelSizeFC2, numChannelsFC2, numOutputChannelsFC2);
    hipDeviceSynchronize();


    hipMemcpy(d_outputFC2, outputFC2.data(), outputFC2.size() * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_biasesFC2, biasesFC2.data(), biasesFC2.size() * sizeof(float), hipMemcpyHostToDevice);

    addBiasesKernel<<<numBlocksFC2, threadsPerBlock>>>(d_outputFC2, d_biasesFC2, outputWidthFC2, outputHeightFC2, numOutputChannelsFC2);


    hipMemcpy(outputFC2.data(), d_outputFC2, outputFC2.size() * sizeof(float), hipMemcpyDeviceToHost);


    softmaxKernel<<<1, threadsPerBlockSoftMax, sharedDataSize>>>(d_outputFC2, d_softmaxOutput, numElementsFC2);
    hipDeviceSynchronize();


    hipMemcpy(softmaxOutput.data(), d_softmaxOutput, numElementsFC2 * sizeof(float), hipMemcpyDeviceToHost);

    // print the softmax output
    // std::cout << "Softmax Output:" << std::endl;
    // for (int i = 0; i < numElementsFC2; ++i) {
    //     std::cout << softmaxOutput[i] << " ";
    // }

    // Find top 5 probabilities and their class indices
    std::vector<std::pair<float, int>> probabilities;
    for (int i = 0; i < numElementsFC2; ++i) {
        probabilities.emplace_back(softmaxOutput[i], i);
    }
    std::sort(probabilities.rbegin(), probabilities.rend()); // Sort in descending order

    // std::cout << "Top 5 Softmax Probabilities:" << std::endl;
    // for (int i = 0; i < 5; ++i) {
    //     std::cout << probabilities[i].first * 100 << "% class " << probabilities[i].second << std::endl;
    // }

    // Print into file
    std::ofstream myfile;
    std::string outpath = "output/" + filename;
    myfile.open(outpath);
    myfile << "Top 5 Softmax Probabilities:" << std::endl;
    for (int i = 0; i < 5; ++i) {
        myfile << probabilities[i].first * 100 << "% class " << probabilities[i].second << std::endl;
    }

    // std::cout << filePath.filename() << " " << probabilities[0].second << std::endl;

        
            }
            }
        }
    }

        // End clock
    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> elapsed = end - start;
    // std::cout << "Time taken: " << elapsed.count() << " s" << std::endl;

    hipFree(d_pooledInputConv1);
    hipFree(d_pooledOutputConv1);
    hipFree(d_pooledInputConv2);
    hipFree(d_pooledOutputConv2);
    hipFree(d_outputFC1);
    hipFree(d_biasesFC1);
    hipFree(d_outputFC2);
    hipFree(d_biasesFC2);
    hipFree(d_softmaxOutput);
    


    return 0;
}